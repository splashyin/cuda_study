
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


int main()
{
    int numElements = 16;
    int totalBytes = numElements * sizeof( int );

    int* deviceArray = 0;
    int* hostArray = 0;

    // malloc host array
    hostArray = ( int* )malloc( totalBytes );

    // cudaMalloc device array
    hipMalloc( ( void** )&deviceArray, totalBytes );

    // set zeros for cuda array
    hipMemset( deviceArray, 0, totalBytes );

    // copy content of the device array to the host
    hipMemcpy( hostArray, deviceArray, totalBytes, hipMemcpyDeviceToHost );


    // print elements in host array
    for ( int i = 0; i < numElements; ++i )
    {
        printf( "%d: %d\n", i, hostArray[ i ] );
    }

    // free host memory
    free( hostArray );

    // free device memory
    hipFree( deviceArray );

    return 0;
}